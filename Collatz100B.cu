// CollatzMultiGPU_Ranged.cu

#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <thread>
#include <mutex>
#include <sstream>
#include <algorithm>

// ANSI color codes for colorful output (optional)
const std::string RESET = "\033[0m";
const std::string BOLD_RED = "\033[1;31m";
const std::string BOLD_GREEN = "\033[1;32m";
const std::string BOLD_YELLOW = "\033[1;33m";
const std::string BOLD_BLUE = "\033[1;34m";

// Structure to represent 128-bit unsigned integers
struct uint128 {
    unsigned long long low;
    unsigned long long high;
};

// Host and device functions for 128-bit arithmetic

__host__ __device__ uint128 add_uint128(uint128 a, uint128 b) {
    uint128 result;
    result.low = a.low + b.low;
    result.high = a.high + b.high + (result.low < a.low ? 1ULL : 0ULL);
    return result;
}

__host__ __device__ uint128 subtract_uint128(uint128 a, uint128 b) {
    uint128 result;
    result.low = a.low - b.low;
    result.high = a.high - b.high - (a.low < b.low ? 1ULL : 0ULL);
    return result;
}

__host__ __device__ uint128 right_shift_uint128(uint128 a) {
    uint128 result;
    result.low = (a.low >> 1) | (a.high << 63);
    result.high = a.high >> 1;
    return result;
}

__host__ __device__ bool is_even_uint128(uint128 a) {
    return (a.low & 1ULL) == 0ULL;
}

__host__ __device__ bool less_than_uint128(uint128 a, uint128 b) {
    return (a.high < b.high) || (a.high == b.high && a.low < b.low);
}

__host__ __device__ bool less_than_or_equal_uint128(uint128 a, uint128 b) {
    return (a.high < b.high) || (a.high == b.high && a.low <= b.low);
}

__host__ __device__ void increment_uint128(uint128* value, unsigned long long increment) {
    unsigned long long old_low = value->low;
    value->low += increment;
    if (value->low < old_low) {
        value->high += 1;
    }
}

// Modified multiply_uint128 function
__host__ __device__ uint128 multiply_uint128(uint128 a, unsigned long long b) {
    uint128 result = {0ULL, 0ULL};

    unsigned long long a_low = a.low;
    unsigned long long a_high = a.high;

    unsigned long long low = a_low * b;
    unsigned long long high = a_high * b;

    // Check for carry from low to high
    if (low < a_low * b) {
        high += 1ULL;
    }

    result.low = low;
    result.high = high;

    return result;
}

// Function to divide uint128 by a small unsigned int
__host__ uint128 divide_uint128_by_uint32(uint128 dividend, uint32_t divisor) {
    uint128 result = {0ULL, 0ULL};
    uint128 temp = {dividend.low, dividend.high};

    // If high part is zero, we can perform simple division
    if (temp.high == 0) {
        result.low = temp.low / divisor;
        result.high = 0;
    } else {
        // Perform division on 128-bit number
        // Split dividend into two 64-bit parts
        unsigned __int128 dividend_128 = ((__int128)temp.high << 64) | temp.low;
        unsigned __int128 result_128 = dividend_128 / divisor;
        result.low = (unsigned long long)(result_128 & 0xFFFFFFFFFFFFFFFFULL);
        result.high = (unsigned long long)(result_128 >> 64);
    }

    return result;
}

// Function to compute uint128 power of 10
__host__ uint128 uint128_pow10(unsigned int exponent) {
    uint128 result = {1ULL, 0ULL};

    for (unsigned int i = 0; i < exponent; ++i) {
        result = multiply_uint128(result, 10ULL);
    }
    return result;
}

// Simplified print_uint128 function
void print_uint128(uint128 a) {
    if (a.high == 0ULL) {
        std::cout << a.low;
    } else {
        // For large numbers, print high and low parts
        std::cout << a.high << std::setw(20) << std::setfill('0') << a.low;
    }
}

// CUDA error-checking macro
#define cudaCheckError(call)                                    \
    {                                                           \
        hipError_t err = call;                                 \
        if (err != hipSuccess) {                               \
            std::cerr << "CUDA error in " << __FILE__           \
                      << " at line " << __LINE__ << ": "        \
                      << hipGetErrorString(err) << std::endl;  \
            exit(EXIT_FAILURE);                                 \
        }                                                       \
    }

// Atomic function to update the maximum steps
__device__ void update_max_steps(unsigned long long *max_steps, uint128 *number_with_max_steps,
                                 unsigned long long local_steps, uint128 local_number) {
    unsigned long long prev_max_steps = atomicMax(max_steps, local_steps);

    if (local_steps > prev_max_steps) {
        unsigned long long* addr = (unsigned long long*)number_with_max_steps;
        atomicExch(addr, local_number.low);
        atomicExch(addr + 1, local_number.high);
    }
}

// Kernel function to find the number with the maximum Collatz steps in a given range
__global__ void find_max_collatz_steps_in_range(uint128 start, uint128 end,
                                                unsigned long long *d_max_steps,
                                                uint128 *d_number_with_max_steps) {
    unsigned long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long stride = gridDim.x * blockDim.x;

    uint128 current = start;
    increment_uint128(&current, idx);

    unsigned long long local_max_steps = 0;
    uint128 local_number_with_max_steps = current;

    while (less_than_or_equal_uint128(current, end)) {
        uint128 n = current;
        unsigned long long steps = 0;

        // Compute Collatz steps
        while (!(n.low == 1ULL && n.high == 0ULL)) {
            if (is_even_uint128(n)) {
                n = right_shift_uint128(n);
            } else {
                n = add_uint128(multiply_uint128(n, 3ULL), {1ULL, 0ULL});
            }
            steps++;
        }

        if (steps > local_max_steps) {
            local_max_steps = steps;
            local_number_with_max_steps = current;
        }

        increment_uint128(&current, stride);
    }

    // Update global maximum
    update_max_steps(d_max_steps, d_number_with_max_steps, local_max_steps, local_number_with_max_steps);
}

// Host function to perform computation on each GPU
void gpu_compute(int gpu_id, uint128 start, uint128 end,
                 unsigned long long &h_max_steps, uint128 &h_number_with_max_steps) {
    // Set the device for this thread
    cudaCheckError(hipSetDevice(gpu_id));

    // Prepare device memory
    unsigned long long *d_max_steps;
    uint128 *d_number_with_max_steps;

    cudaCheckError(hipMalloc(&d_max_steps, sizeof(unsigned long long)));
    cudaCheckError(hipMalloc(&d_number_with_max_steps, sizeof(uint128)));

    cudaCheckError(hipMemset(d_max_steps, 0, sizeof(unsigned long long)));
    cudaCheckError(hipMemset(d_number_with_max_steps, 0, sizeof(uint128)));

    // Determine optimal block size and grid size using cudaOccupancyMaxPotentialBlockSize
    int minGridSize = 0;
    int blockSize = 0;
    cudaCheckError(hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        find_max_collatz_steps_in_range,
        0,  // dynamic shared memory per block
        0)); // block size limit

    // Compute total number of elements (numbers) in the range
    // Since total_numbers can be very large, we limit the grid size to a reasonable number
    unsigned long long total_numbers = 0;
    if (end.high == start.high) {
        total_numbers = end.low - start.low + 1;
    } else {
        // If the high parts are different, the range is too large to represent in 64 bits
        total_numbers = ~0ULL; // Set to maximum possible value
    }

    int gridSize = (total_numbers + blockSize - 1) / blockSize;

    // Limit gridSize to a maximum value to prevent excessive resource usage
    int maxGridSize = 65535; // Maximum grid size per dimension
    if (gridSize > maxGridSize) {
        gridSize = maxGridSize;
    }

    // Debug: Print GPU ID and assigned range
    std::cout << "GPU " << gpu_id << " processing range: Start = ";
    print_uint128(start);
    std::cout << ", End = ";
    print_uint128(end);
    std::cout << std::endl;

    // Launch kernel
    find_max_collatz_steps_in_range<<<gridSize, blockSize>>>(start, end, d_max_steps, d_number_with_max_steps);

    // Synchronize
    cudaCheckError(hipDeviceSynchronize());

    // Copy results back to host
    cudaCheckError(hipMemcpy(&h_max_steps, d_max_steps, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(&h_number_with_max_steps, d_number_with_max_steps, sizeof(uint128), hipMemcpyDeviceToHost));

    // Free device memory
    cudaCheckError(hipFree(d_max_steps));
    cudaCheckError(hipFree(d_number_with_max_steps));
}

int main() {
    std::cout << "Program started." << std::endl;

    // Get device count
    int device_count = 0;
    cudaCheckError(hipGetDeviceCount(&device_count));

    std::cout << "Number of CUDA devices: " << device_count << std::endl;

    if (device_count < 1) {
        std::cerr << "No CUDA-capable devices found." << std::endl;
        return 1;
    }

    // Generate powers of 10 from 10^0 to 10^n (n <= 38 due to uint128 limitations)
    const unsigned int max_exponent = 20; // Adjust as needed, max 38
    std::vector<uint128> powers_of_10;

    for (unsigned int i = 0; i <= max_exponent + 1; ++i) {
        uint128 power = uint128_pow10(i);
        powers_of_10.push_back(power);
    }

    // Start timing
    auto total_start_time = std::chrono::high_resolution_clock::now();

    // Loop over each range
    for (size_t range_idx = 0; range_idx < powers_of_10.size() - 1; ++range_idx) {
        uint128 start = powers_of_10[range_idx];
        uint128 end = subtract_uint128(powers_of_10[range_idx + 1], {1ULL, 0ULL});

        // Calculate the total range
        uint128 total_range = subtract_uint128(end, start);
        increment_uint128(&total_range, 1ULL); // total_range = end - start + 1

        // Variables to hold the maximum steps and corresponding numbers from each GPU
        std::vector<unsigned long long> max_steps_per_gpu(device_count, 0);
        std::vector<uint128> number_with_max_steps_per_gpu(device_count);

        std::cout << BOLD_BLUE << "Processing range: ";
        print_uint128(start);
        std::cout << " to ";
        print_uint128(end);
        std::cout << RESET << std::endl;

        // Start timing for this range
        auto range_start_time = std::chrono::high_resolution_clock::now();

        // Split the total range among GPUs
        std::vector<std::pair<uint128, uint128>> gpu_ranges(device_count);

        // Calculate the size of each subrange
        uint128 one = {1ULL, 0ULL};

        for (int i = 0; i < device_count; ++i) {
            // Calculate start offset
            uint128 index = {static_cast<unsigned long long>(i), 0ULL};
            uint128 range_offset = multiply_uint128(total_range, index.low);
            range_offset = divide_uint128_by_uint32(range_offset, device_count);

            gpu_ranges[i].first = add_uint128(start, range_offset);

            // Calculate end offset
            uint128 next_index = {static_cast<unsigned long long>(i + 1), 0ULL};
            uint128 next_range_offset = multiply_uint128(total_range, next_index.low);
            next_range_offset = divide_uint128_by_uint32(next_range_offset, device_count);

            gpu_ranges[i].second = subtract_uint128(add_uint128(start, next_range_offset), one);

            // Ensure that the last GPU's end range is correct
            if (i == device_count - 1) {
                gpu_ranges[i].second = end;
            }
        }

        // Create and start threads for each GPU
        std::vector<std::thread> gpu_threads(device_count);
        for (int i = 0; i < device_count; ++i) {
            gpu_threads[i] = std::thread([&, i]() {
                // Each thread sets its own device
                cudaCheckError(hipSetDevice(i));
                gpu_compute(i, gpu_ranges[i].first, gpu_ranges[i].second,
                            std::ref(max_steps_per_gpu[i]),
                            std::ref(number_with_max_steps_per_gpu[i]));
            });
        }

        // Wait for all threads to finish
        for (int i = 0; i < device_count; ++i) {
            gpu_threads[i].join();
        }

        std::cout << "GPU computations completed for this range." << std::endl;

        // Find the overall maximum steps and corresponding number
        unsigned long long h_max_steps = 0;
        uint128 h_number_with_max_steps = {0ULL, 0ULL};
        for (int i = 0; i < device_count; ++i) {
            if (max_steps_per_gpu[i] > h_max_steps) {
                h_max_steps = max_steps_per_gpu[i];
                h_number_with_max_steps = number_with_max_steps_per_gpu[i];
            }
        }

        // End timing for this range
        auto range_end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> execution_time = range_end_time - range_start_time;

        // Display the result for this range
        std::cout << BOLD_GREEN << "Number with max steps: ";
        print_uint128(h_number_with_max_steps);
        std::cout << RESET << std::endl;

        std::cout << BOLD_YELLOW << "Steps: " << h_max_steps << RESET << std::endl;
        std::cout << BOLD_RED << "Time taken for this range: " << execution_time.count() << " seconds" << RESET << std::endl;
        std::cout << "-----------------------------" << std::endl;
    }

    // End timing
    auto total_end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> total_execution_time = total_end_time - total_start_time;

    std::cout << BOLD_RED << "Total time taken: " << total_execution_time.count() << " seconds" << RESET << std::endl;

    return 0;
}
