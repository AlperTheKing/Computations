// CollatzMultiGPU_Ranged.cu

#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <thread>
#include <mutex>
#include <sstream>
#include <algorithm>

// ANSI color codes for colorful output (optional)
const std::string RESET = "\033[0m";
const std::string BOLD_RED = "\033[1;31m";
const std::string BOLD_GREEN = "\033[1;32m";
const std::string BOLD_YELLOW = "\033[1;33m";
const std::string BOLD_BLUE = "\033[1;34m";

// Structure to represent 128-bit unsigned integers
struct uint128 {
    unsigned long long low;
    unsigned long long high;
};

// Host and device functions for 128-bit arithmetic

__host__ __device__ uint128 add_uint128(uint128 a, uint128 b) {
    uint128 result;
    result.low = a.low + b.low;
    result.high = a.high + b.high + (result.low < a.low ? 1ULL : 0ULL);
    return result;
}

__host__ __device__ uint128 subtract_uint128(uint128 a, uint128 b) {
    uint128 result;
    result.low = a.low - b.low;
    result.high = a.high - b.high - (a.low < b.low ? 1ULL : 0ULL);
    return result;
}

__host__ __device__ uint128 right_shift_uint128(uint128 a) {
    uint128 result;
    result.low = (a.low >> 1) | (a.high << 63);
    result.high = a.high >> 1;
    return result;
}

__host__ __device__ bool is_even_uint128(uint128 a) {
    return (a.low & 1ULL) == 0ULL;
}

__host__ __device__ bool less_than_uint128(uint128 a, uint128 b) {
    return (a.high < b.high) || (a.high == b.high && a.low < b.low);
}

__host__ __device__ bool less_than_or_equal_uint128(uint128 a, uint128 b) {
    return (a.high < b.high) || (a.high == b.high && a.low <= b.low);
}

__host__ __device__ void increment_uint128(uint128* value, unsigned long long increment) {
    unsigned long long old_low = value->low;
    value->low += increment;
    if (value->low < old_low) {
        value->high += 1;
    }
}

// Modified multiply_uint128 function
__host__ __device__ uint128 multiply_uint128(uint128 a, unsigned long long b) {
    uint128 result = {0ULL, 0ULL};

    unsigned long long a_low = a.low;
    unsigned long long a_high = a.high;

    unsigned int al_low = (unsigned int)(a_low & 0xFFFFFFFFULL);
    unsigned int al_high = (unsigned int)(a_low >> 32);

    unsigned int b_low = (unsigned int)(b & 0xFFFFFFFFULL);
    unsigned int b_high = (unsigned int)(b >> 32);

    // Compute partial products
    unsigned long long ll = (unsigned long long)al_low * b_low;
    unsigned long long lh = (unsigned long long)al_low * b_high;
    unsigned long long hl = (unsigned long long)al_high * b_low;
    unsigned long long hh = (unsigned long long)al_high * b_high;

    // Combine partial products
    unsigned long long mid = lh + hl;
    unsigned long long carry = (mid < lh) ? (1ULL << 32) : 0ULL;

    result.low = ll + (mid << 32);
    if (result.low < ll) carry++;

    result.high = a_high * b + hh + (mid >> 32) + carry;

    return result;
}

// Function to divide uint128 by an unsigned int
__host__ uint128 divide_uint128(uint128 dividend, unsigned int divisor) {
    uint128 result = {0ULL, 0ULL};
    uint128 remainder = {0ULL, 0ULL};

    for (int i = 127; i >= 0; --i) {
        // Left shift remainder by 1
        remainder.high = (remainder.high << 1) | (remainder.low >> 63);
        remainder.low = (remainder.low << 1);

        // Bring down the next bit of the dividend
        if (i >= 64) {
            remainder.low |= (dividend.high >> (i - 64)) & 1ULL;
        } else {
            remainder.low |= (dividend.low >> i) & 1ULL;
        }

        // If remainder >= divisor
        if (remainder.high > 0 || remainder.low >= divisor) {
            remainder.low -= divisor;
            // Set the corresponding bit in the result
            if (i >= 64) {
                result.high |= (1ULL << (i - 64));
            } else {
                result.low |= (1ULL << i);
            }
        }
    }
    return result;
}

// Function to compute uint128 power of 10
__host__ uint128 uint128_pow10(unsigned int exponent) {
    uint128 result = {1ULL, 0ULL};

    for (unsigned int i = 0; i < exponent; ++i) {
        result = multiply_uint128(result, 10ULL);
    }
    return result;
}

// Simplified print_uint128 function
void print_uint128(uint128 a) {
    if (a.high == 0ULL) {
        std::cout << a.low;
    } else {
        // For large numbers, print high and low parts
        std::cout << a.high << std::setw(20) << std::setfill('0') << a.low;
    }
}

// CUDA error-checking macro
#define cudaCheckError(call)                                    \
    {                                                           \
        hipError_t err = call;                                 \
        if (err != hipSuccess) {                               \
            std::cerr << "CUDA error in " << __FILE__           \
                      << " at line " << __LINE__ << ": "        \
                      << hipGetErrorString(err) << std::endl;  \
            exit(EXIT_FAILURE);                                 \
        }                                                       \
    }

// Atomic function to update the maximum steps
__device__ void update_max_steps(unsigned long long *max_steps, uint128 *number_with_max_steps,
                                 unsigned long long local_steps, uint128 local_number) {
    unsigned long long prev_max_steps = atomicMax(max_steps, local_steps);

    if (local_steps > prev_max_steps) {
        unsigned long long* addr = (unsigned long long*)number_with_max_steps;
        atomicExch(addr, local_number.low);
        atomicExch(addr + 1, local_number.high);
    }
}

// Kernel function to find the number with the maximum Collatz steps in a given range
__global__ void find_max_collatz_steps_in_range(uint128 start, uint128 end,
                                                unsigned long long *d_max_steps,
                                                uint128 *d_number_with_max_steps) {
    unsigned long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long stride = gridDim.x * blockDim.x;

    uint128 current = start;
    increment_uint128(&current, idx);

    unsigned long long local_max_steps = 0;
    uint128 local_number_with_max_steps = current;

    while (less_than_or_equal_uint128(current, end)) {
        uint128 n = current;
        unsigned long long steps = 0;

        // Compute Collatz steps
        while (!(n.low == 1ULL && n.high == 0ULL)) {
            if (is_even_uint128(n)) {
                n = right_shift_uint128(n);
            } else {
                n = add_uint128(multiply_uint128(n, 3ULL), {1ULL, 0ULL});
            }
            steps++;
        }

        if (steps > local_max_steps) {
            local_max_steps = steps;
            local_number_with_max_steps = current;
        }

        increment_uint128(&current, stride);
    }

    // Update global maximum
    update_max_steps(d_max_steps, d_number_with_max_steps, local_max_steps, local_number_with_max_steps);
}

// Host function to perform computation on each GPU
void gpu_compute(int gpu_id, uint128 start, uint128 end,
                 unsigned long long &h_max_steps, uint128 &h_number_with_max_steps) {
    // Set the device for this thread
    cudaCheckError(hipSetDevice(gpu_id));

    // Prepare device memory
    unsigned long long *d_max_steps;
    uint128 *d_number_with_max_steps;

    cudaCheckError(hipMalloc(&d_max_steps, sizeof(unsigned long long)));
    cudaCheckError(hipMalloc(&d_number_with_max_steps, sizeof(uint128)));

    cudaCheckError(hipMemset(d_max_steps, 0, sizeof(unsigned long long)));
    cudaCheckError(hipMemset(d_number_with_max_steps, 0, sizeof(uint128)));

    // Determine optimal block size and grid size
    int device;
    hipDeviceProp_t deviceProp;
    cudaCheckError(hipGetDevice(&device));
    cudaCheckError(hipGetDeviceProperties(&deviceProp, device));

    int blockSize = 256; // Adjust based on your GPU
    int gridSize = (deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount) / blockSize;

    // Debug: Print GPU ID and assigned range
    std::cout << "GPU " << gpu_id << " processing range: Start = ";
    print_uint128(start);
    std::cout << ", End = ";
    print_uint128(end);
    std::cout << std::endl;

    // Launch kernel
    find_max_collatz_steps_in_range<<<gridSize, blockSize>>>(start, end, d_max_steps, d_number_with_max_steps);

    // Synchronize
    cudaCheckError(hipDeviceSynchronize());

    // Copy results back to host
    cudaCheckError(hipMemcpy(&h_max_steps, d_max_steps, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(&h_number_with_max_steps, d_number_with_max_steps, sizeof(uint128), hipMemcpyDeviceToHost));

    // Free device memory
    cudaCheckError(hipFree(d_max_steps));
    cudaCheckError(hipFree(d_number_with_max_steps));
}

int main() {
    std::cout << "Program started." << std::endl;

    // Get device count
    int device_count = 0;
    cudaCheckError(hipGetDeviceCount(&device_count));

    std::cout << "Number of CUDA devices: " << device_count << std::endl;

    if (device_count < 1) {
        std::cerr << "No CUDA-capable devices found." << std::endl;
        return 1;
    }

    // Generate powers of 10 from 10^0 to 10^n (n <= 38 due to uint128 limitations)
    const unsigned int max_exponent = 20; // Adjust as needed, max 38
    std::vector<uint128> powers_of_10;

    for (unsigned int i = 0; i <= max_exponent + 1; ++i) {
        uint128 power = uint128_pow10(i);
        powers_of_10.push_back(power);
    }

    // Start timing
    auto total_start_time = std::chrono::high_resolution_clock::now();

    // Loop over each range
    for (size_t range_idx = 0; range_idx < powers_of_10.size() - 1; ++range_idx) {
        uint128 start = powers_of_10[range_idx];
        uint128 end = subtract_uint128(powers_of_10[range_idx + 1], {1ULL, 0ULL});

        // Calculate the total range
        uint128 total_range = subtract_uint128(end, start);
        increment_uint128(&total_range, 1ULL); // total_range = end - start + 1

        // Split the total range among GPUs
        std::vector<std::pair<uint128, uint128>> gpu_ranges(device_count);

        // Calculate the size of each subrange
        uint128 range_size = divide_uint128(total_range, device_count);

        uint128 one = {1ULL, 0ULL};

        for (int i = 0; i < device_count; ++i) {
            gpu_ranges[i].first = add_uint128(start, multiply_uint128(range_size, i));
            if (i == device_count - 1) {
                gpu_ranges[i].second = end;
            } else {
                gpu_ranges[i].second = subtract_uint128(add_uint128(gpu_ranges[i].first, range_size), one);
            }
        }

        // Variables to hold the maximum steps and corresponding numbers from each GPU
        std::vector<unsigned long long> max_steps_per_gpu(device_count, 0);
        std::vector<uint128> number_with_max_steps_per_gpu(device_count);

        std::cout << BOLD_BLUE << "Processing range: ";
        print_uint128(start);
        std::cout << " to ";
        print_uint128(end);
        std::cout << RESET << std::endl;

        // Start timing for this range
        auto range_start_time = std::chrono::high_resolution_clock::now();

        // Create and start threads for each GPU
        std::vector<std::thread> gpu_threads(device_count);
        for (int i = 0; i < device_count; ++i) {
            gpu_threads[i] = std::thread([&, i]() {
                // Each thread sets its own device
                cudaCheckError(hipSetDevice(i));
                gpu_compute(i, gpu_ranges[i].first, gpu_ranges[i].second,
                            std::ref(max_steps_per_gpu[i]),
                            std::ref(number_with_max_steps_per_gpu[i]));
            });
        }

        // Wait for all threads to finish
        for (int i = 0; i < device_count; ++i) {
            gpu_threads[i].join();
        }

        std::cout << "GPU computations completed for this range." << std::endl;

        // Find the overall maximum steps and corresponding number
        unsigned long long h_max_steps = 0;
        uint128 h_number_with_max_steps = {0ULL, 0ULL};
        for (int i = 0; i < device_count; ++i) {
            if (max_steps_per_gpu[i] > h_max_steps) {
                h_max_steps = max_steps_per_gpu[i];
                h_number_with_max_steps = number_with_max_steps_per_gpu[i];
            }
        }

        // End timing for this range
        auto range_end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> execution_time = range_end_time - range_start_time;

        // Display the result for this range
        std::cout << BOLD_GREEN << "Number with max steps: ";
        print_uint128(h_number_with_max_steps);
        std::cout << RESET << std::endl;

        std::cout << BOLD_YELLOW << "Steps: " << h_max_steps << RESET << std::endl;
        std::cout << BOLD_RED << "Time taken for this range: " << execution_time.count() << " seconds" << RESET << std::endl;
        std::cout << "-----------------------------" << std::endl;
    }

    // End timing
    auto total_end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> total_execution_time = total_end_time - total_start_time;

    std::cout << BOLD_RED << "Total time taken: " << total_execution_time.count() << " seconds" << RESET << std::endl;

    return 0;
}
